#include "hip/hip_runtime.h"
#pragma once

#include <stdexcept>

/*
 * Quotient and remainder via Barrett reduction.
 *
 * div: the divisor
 * (mu, mu_msw): floor(2^(2*NBITS) / div) where NBITS = FIXNUM_BITS
 * width: digits in div and mu.
 */
template< typename fixnum_impl >
class quorem {
public:
    typedef typename fixnum_impl::fixnum fixnum;

    // TODO: mu and mu_msw should be calculated from div on the device
    // when we support general modular inverses.
    __device__ quorem(fixnum div, fixnum mu, fixnum mu_msw);

    __device__ void operator()(fixnum &q, fixnum &r, fixnum A_hi, fixnum A_lo) const;

    // Just return the remainder.
    __device__ void operator()(fixnum &r, fixnum A_hi, fixnum A_lo) const {
        fixnum q;
        (*this)(q, r, A_hi, A_lo);
    }

private:
    static constexpr int fixnum_impl::SLOT_WIDTH WIDTH;

    fixnum div, mu, mu_msw;
};

/*
 * Create a quorem object.
 *
 * Throws an exception if div does not have a sufficiently high bit switched
 * on, or if nbytes > FIXNUM_BYTES.
 */
template< typename fixnum_impl >
__device__
quorem<fixnum_impl>::quorem(fixnum div_, fixnum mu_, fixnum mu_msw_)
    : div(div_), mu(mu_), mu_msw(mu_msw_)
{
    // Require at least one of the high 4 bits to be switched on. This
    // determines the maximum number of corrections needed at the end
    // of a reduction.
    static constexpr word_tp MIN_MSW = (word_tp)1 << (WORD_BITS - 5);

    // FIXME: This is not the right way to enforce the restriction on
    // the relative sizes of the divisor and the dividend.
    assert(fixnum_impl::most_sig_dig(div) < MIN_MSW);
}


/*
 * Return the quotient and remainder of A after division by div.
 *
 * Uses Barret reduction.  See HAC, Algo 14.42, and MCA, Algo 2.5.
 *
 * TODO: Explain how this implementation deviates from the algorithms
 * cited above, in particular how it relates to the expected number of
 * iterations of the "correction loop".  NB: It's possible this
 * "deviation" can be removed by "normalising" the relevant data (see
 * MCA, Section 1.4.1).
 */
template< typename fixnum_impl >
__device__ void
quorem<fixnum_impl>::operator()(fixnum &q, fixnum &r, fixnum A_hi, fixnum A_lo) const
{
    fixnum t, msw, hi, lo, br;

    int L = slot_layout::laneIdx();

    // (q, msw) = "A_hi * mu / 2^NBITS"
    // TODO: the lower half of the product, t, is unused, so we might
    // be able to use a mul_hi() function that only calculates an
    // approximate answer (see Short Product discussion at MCA,
    // Section 3.3 (from Section 2.4.1, p59)).
    fixnum_impl::mul_wide(q, t, A_hi, mu);
    msw = fixnum_impl::mad_cy(q, A_hi, mu_msw);

    // (hi, lo) = q*d
    fixnum_impl::mul_wide(hi, lo, q, div);
    msw = fixnum_impl::mad_cy(hi, div, msw);

    // q*d always fits in two fixnums, even though msw of q is
    // sometimes non-zero.
    assert(msw == 0);

    // (r, msw) = A - q*d
    br = fixnum_impl::sub_br(r, A_lo, lo);
    t = fixnum_impl::sub_br(msw, A_hi, hi);

    // A_hi >= hi
    assert(t == 0);

    // make br into a fixnum
    // FIXME: check why I can't just do "msw -= br" here.
    br = (L == 0) ? br : 0;
    t = fixnum_impl::sub_br(msw, msw, br);

    // msw >= br
    assert(t == 0);
    // msw < 2^64
    assert(L == 0 || msw == 0);
    msw = slot_layout::shfl(msw, 0);

    // NB: Could call incr_cy in the loops instead; as is it will
    // incur an extra add_cy even when msw is 0 and r < d.
    fixnum q_inc = 0;
    while (msw) {
        msw -= fixnum_impl::sub_br(r, r, div);
        ++q_inc;
    }
    while ( ! fixnum_impl::sub_br(t, r, div)) {
        r = t;
        ++q_inc;
    }
    q_inc = (L == 0) ? q_inc : 0;
    fixnum_impl::add_cy(q, q, q_inc);
}

