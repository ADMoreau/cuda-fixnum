#include "hip/hip_runtime.h"
// -*- compile-command: "nvcc -D__STRICT_ANSI__ -ccbin clang-3.8 -Xcompiler -Wall,-Wextra -g -G -lineinfo -gencode arch=compute_50,code=sm_50 -o bench bench.cu -lstdc++" -*-

#include <iostream>
#include <cstring>

#include "cuda_wrap.h"
#include "hand.cu"

using namespace std;

// parameterised by
// hand implementation, which determines #bits per fixnum
//   and which is itself parameterised by
// subwarp data, which determines a SIMD decomposition of a fixnum
//
// TODO: Copy over functionality and documentation from IntmodVector.
template< typename hand_impl >
class fixnum_array {
public:
    template< typename T >
    static fixnum_array *create(size_t nelts, T init = 0) {
        fixnum_array *a;
        cuda_malloc_managed(&a, sizeof(*a));
        a->nelts = nelts;
        if (nelts > 0) {
            size_t nbytes = nelts * hand_impl::FIXNUM_BYTES;
            int c = static_cast<int>(init);
            cuda_malloc(&a->ptr, nbytes);
            // FIXME: Obviously should use zeros and init somehow
            cuda_memset(a->ptr, c, nbytes);
        }
    }

    static fixnum_array *create(const uint8_t *data, size_t len, size_t bytes_per_elt);

    // TODO: Any advantage to making this a normal method "destroy()"?
    ~fixnum_array() {
        if (a->nelts > 0)
            cuda_free(a->ptr);
        cuda_free(a);
    }

    void retrieve_into(uint8_t *dest, size_t dest_space, size_t *res_len, int idx) {
        size_t nbytes = hand_impl::FIXNUM_BYTES;
        if (dest_space < nbytes || idx < 0 || idx > nelts) {
            // FIXME: This is not the right way to handle an
            // "insufficient space" error or an "index out of bounds"
            // error.
            *res_len = 0;
            return;
        }
        // clear all of dest
        // TODO: Is this necessary? Should it be optional?
        memset(dest, 0, dest_space);
        cuda_memcpy_from_device(dest, a->ptr + idx * nbytes, nbytes);
        *res_len = nbytes;
    }

    void retrieve(uint8_t **dest, size_t *dest_len, int idx) {
        *dest_len = hand_impl::FIXNUM_BYTES;
        *dest = new uint8_t[*dest_len];
        retrieve_into(&dest, *dest_len, dest_len, idx);
    }

    void retrieve_all(uint8_t **dest, size_t *dest_len, size_t *nelts) {
        size_t nbytes;
        *nelts = this->nelts;
        nbytes = *nelts * hand_impl::FIXNUM_BYTES;
        *dest = new uint8_t[nbytes];
        // FIXME: This won't correctly zero-pad each element
        memset(dest, 0, nbytes);
        cuda_memcpy_from_device(*dest, a->ptr, nbytes);
    }
#if 0
    int add_cy(const fixnum_array *other) {
        // FIXME: Return correct carry
        int cy = 0;
        apply_to_all(hand_impl::add_cy, this, other);
        return cy;
    }

    void mullo(const fixnum_array *other) {
        apply_to_all(hand_impl::mullo, this, other);
    }
#endif

private:
    // FIXME: This shouldn't be public; the create function that uses
    // it should be templatised.
    typedef typename hand_impl::digit value_tp;

    value_tp *ptr;
    int nelts;

    fixnum_array();
    fixnum_array(const fixnum_array &);
    fixnum_array &operator=(const fixnum_array &);

#if 0
    template< typename Func >
    __global__ void
    binary_dispatch(const Func *fn, fixnum_array *dest,
            const fixnum_array *src1, const fixnum_array *src2) {
        int blk_tid_offset = blockDim.x * blockIdx.x;
        int tid_in_blk = threadIdx.x;
        int fn_idx = (blk_tid_offset + tid_in_blk) / fn_width;

        if (fn_idx < Z->nelts) {
            int zoff = fn_idx * Z->width;
            int xoff = fn_idx * X->width;
            fn->apply(Z->ptr + zoff, Z->width, X->ptr + xoff, X->width);
        }
    }

    // TODO: Set this to the number of threads on a single SM on the host GPU.
    template< typename Func, int BLOCK_SIZE = 192 >
    void
    apply_to_all(const Func *fn, fixnum_array *dest, const fixnum_array *src, clock_t *t) {
        // dest and src must be the same length
        assert(dest->nelts == src->nelts);
        // BLOCK_SIZE must be a multiple of warpSize
        static_assert(!(BLOCK_SIZE % warpSize));

        // FIXME: Check this calculation
        //int fixnums_per_block = (BLOCK_SIZE / warpSize) * hand_impl::NSLOTS;
        int fixnums_per_block = BLOCK_SIZE / hand_impl::SLOT_WIDTH;

        // FIXME: nblocks could be too big for a single kernel call to handle
        int nblocks = iceil(src->nelts, fixnums_per_block);

        if (t) *t = clock();
        // nblocks > 0 iff src->nelts > 0
        if (nblocks > 0) {
            hipStream_t stream;
            cuda_check(hipStreamCreate(&stream), "create stream (binary dispatch)");
            // FIXME: how do I attach the function?
            //stream_attach(stream, fn);
            cuda_stream_attach_mem(stream, src);
            cuda_stream_attach_mem(stream, dest);
            cuda_check(hipStreamSynchronize(stream), "stream sync");

            binary_dispatch<<< nblocks, BLOCK_SIZE, 0, stream >>>(fn, dest, src);

            cuda_check(hipPeekAtLastError(), "kernel invocation/run");
            cuda_check(hipStreamSynchronize(stream), "stream sync");
            cuda_check(hipStreamDestroy(stream), "stream destroy");
        }
        if (t) *t = clock() - *t;
    }
#endif
};


// FIXME: Ignore this idea of feeding in new operations for now; just
// use a fixed set of operations determined by hand_impl
// 
// FIXME: Passing this to map as an object probably makes inlining
// impossible in most circumstances.
template< typename T, typename subwarp_data >
struct device_op {
    int _x;

    device_op(int x) : _x(x) { }

    // A fixnum is represented by a register across a subwarp. This
    // thread is responsible for the Lth registers of the arguments,
    // where L is the lane index.
    //
    // This function should be available from the hand_impl; this sort
    // of function should be implemented in terms of the hand_impl
    // functions.
    __device__ void
    operator()(T &s, T &cy, T a, T b) {
        s = a + b;
        cy = s < a;
    }
};


template< typename U >
ostream &
operator<<(ostream &os, const fixnum_array<U> &arr) {
    os << "( " << arr[0];
    for (int i = 1; i < N; ++i)
        os << ", " << arr[i];
    os << " )" << flush;
    return os;
}


int main(int argc, char *argv[]) {
    long n = 16;
    if (argc > 1)
        n = atol(argv[1]);

    // hand_impl determines how operations map to a warp
    //
    // bits_per_fixnum should somehow capture the fact that a warp can
    // be divided into subwarps
    //
    // n is the number of fixnums in the array; eventually only allow
    // initialisation via a byte array or whatever
    typedef fixnum_array< full_hand<uint32_t, 4> > fixnum_array;
    auto arr1 = fixnum_array::create(n);
    auto arr2 = fixnum_array::create(n);

    // device_op should be able to start operating on the appropriate
    // memory straight away
    device_op fn(7);

    // FIXME: How do I return cy without allocating a gigantic array
    // where each element is only 0 or 1?  Could return the carries in
    // the device_op fn?
    //fixnum_array::map(fn, res, arr1, arr2);

    cout << "arr1 = " << arr1 << endl;

    delete arr1;
    delete arr2;

    return 0;
}
