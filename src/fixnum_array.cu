#include "hip/hip_runtime.h"
#include "cuda_wrap.h"
#include "hand.cu"

#include "fixnum_array.h"

// TODO: Passing both Hand and Func is ugly; Hand should be implicit
// in Func.
template< typename Func, typename... Args >
__global__ void
dispatch(Func fn, int nelts, Args... args) {

    // TODO NEXT: This stuff should go in some pre/post-hook code in
    // some parent class for device functions
    
    int blk_tid_offset = blockDim.x * blockIdx.x;
    int tid_in_blk = threadIdx.x;
    int fn_idx = (blk_tid_offset + tid_in_blk) / Hand::SLOT_WIDTH;

    if (fn_idx < nelts) {
        //dest->ptr + off, dest->ptr + off, src->ptr + off);
        Hand::call(fn, fn_idx, args...);
    }

    // NEXT: just use this
    
    fn(nelts, args...);
}

template< typename hand_impl >
template< typename T >
fixnum_array<hand_impl> *
fixnum_array<hand_impl>::create(size_t nelts, T init) {
    fixnum_array *a = new fixnum_array;
    a->nelts = nelts;
    if (nelts > 0) {
        size_t nbytes = nelts * hand_impl::FIXNUM_BYTES;
        int c = static_cast<int>(init);
        cuda_malloc(&a->ptr, nbytes);
        // FIXME: Obviously should use zeros and init somehow
        cuda_memset(a->ptr, c, nbytes);
    }
    return a;
}


template< typename hand_impl >
fixnum_array<hand_impl>::~fixnum_array() {
    if (nelts > 0)
        cuda_free(ptr);
}

template< typename hand_impl >
int
fixnum_array<hand_impl>::length() const {
    return nelts;
}

template< typename hand_impl >
size_t
fixnum_array<hand_impl>::retrieve_into(uint8_t *dest, size_t dest_space, int idx) const {
    size_t nbytes = hand_impl::FIXNUM_BYTES;
    if (dest_space < nbytes || idx < 0 || idx > nelts) {
        // FIXME: This is not the right way to handle an
        // "insufficient space" error or an "index out of bounds"
        // error.
        return 0;
    }
    // clear all of dest
    // TODO: Is this necessary? Should it be optional?
    memset(dest, 0, dest_space);
    cuda_memcpy_from_device(dest, ptr + idx * hand_impl::SLOT_WIDTH, nbytes);
    return nbytes;
}

template< typename hand_impl >
void
fixnum_array<hand_impl>::retrieve(uint8_t **dest, size_t *dest_len, int idx) const {
    *dest_len = hand_impl::FIXNUM_BYTES;
    *dest = new uint8_t[*dest_len];
    retrieve_into(*dest, *dest_len, idx);
}

template< typename hand_impl >
void
fixnum_array<hand_impl>::retrieve_all(uint8_t **dest, size_t *dest_len, size_t *nelts) const {
    size_t nbytes;
    *nelts = this->nelts;
    nbytes = *nelts * hand_impl::FIXNUM_BYTES;
    *dest = new uint8_t[nbytes];
    // FIXME: This won't correctly zero-pad each element
    memset(dest, 0, nbytes);
    cuda_memcpy_from_device(*dest, ptr, nbytes);
}

// TODO: Currently restricted to a single return value of type
// fixnum_array. We might want to return multiple values, possible of
// plain arrays (e.g. add_cy could return a fixnum_array and an int
// array).
template< typename Func, typename... Args >
void
mapcar(Func fn, Args... args) {
    // TODO: Set this to the number of threads on a single SM on the host GPU.
    constexpr int BLOCK_SIZE = 192;

    // BLOCK_SIZE must be a multiple of warpSize
    static_assert(!(BLOCK_SIZE % WARPSIZE),
            "block size must be a multiple of warpSize");

    // FIXME: check that arrays are all the same length. Or find the minimum
    // length and use that?
    //assert(nelts == src->nelts);

    // FIXME: Check this calculation
    //int fixnums_per_block = (BLOCK_SIZE / warpSize) * hand_impl::NSLOTS;
    constexpr int fixnums_per_block = BLOCK_SIZE / hand_impl::SLOT_WIDTH;

    // FIXME: nblocks could be too big for a single kernel call to handle
    int nblocks = iceil(src->nelts, fixnums_per_block);

    if (t) *t = clock();
    // nblocks > 0 iff src->nelts > 0
    if (nblocks > 0) {
        hipStream_t stream;
        cuda_check(hipStreamCreate(&stream), "create stream");
        // FIXME: how do I attach the function?
        //stream_attach(stream, fn);
//         cuda_stream_attach_mem(stream, src->ptr);
//         cuda_stream_attach_mem(stream, ptr);
        cuda_check(hipStreamSynchronize(stream), "stream sync");

        dispatch<<< nblocks, BLOCK_SIZE, 0, stream >>>(fn, nelts, args...);

        cuda_check(hipPeekAtLastError(), "kernel invocation/run");
        cuda_check(hipStreamSynchronize(stream), "stream sync");
        cuda_check(hipStreamDestroy(stream), "stream destroy");
    }
    if (t) *t = clock() - *t;
}
