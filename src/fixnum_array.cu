#include "hip/hip_runtime.h"
#include "cuda_wrap.h"
#include "hand.cu"

#include "fixnum_array.h"

// TODO: Passing both Hand and Func is ugly; Hand should be implicit
// in Func.
template< typename Hand, typename Func, typename... Args >
__global__ void
dispatch(Func fn, int nelts, Args... args) {
    int blk_tid_offset = blockDim.x * blockIdx.x;
    int tid_in_blk = threadIdx.x;
    int fn_idx = (blk_tid_offset + tid_in_blk) / Hand::SLOT_WIDTH;

    if (fn_idx < nelts) {
        int off = fn_idx * Hand::SLOT_WIDTH;

        //dest->ptr + off, dest->ptr + off, src->ptr + off);
        fn(off, args...);
    }
}

template< typename hand_impl >
template< typename T >
fixnum_array<hand_impl> *
fixnum_array<hand_impl>::create(size_t nelts, T init) {
    fixnum_array *a = new fixnum_array;
    a->nelts = nelts;
    if (nelts > 0) {
        size_t nbytes = nelts * hand_impl::FIXNUM_BYTES;
        int c = static_cast<int>(init);
        cuda_malloc(&a->ptr, nbytes);
        // FIXME: Obviously should use zeros and init somehow
        cuda_memset(a->ptr, c, nbytes);
    }
    return a;
}


template< typename hand_impl >
fixnum_array<hand_impl>::~fixnum_array() {
    if (nelts > 0)
        cuda_free(ptr);
}

template< typename hand_impl >
int
fixnum_array<hand_impl>::length() const {
    return nelts;
}

template< typename hand_impl >
size_t
fixnum_array<hand_impl>::retrieve_into(uint8_t *dest, size_t dest_space, int idx) const {
    size_t nbytes = hand_impl::FIXNUM_BYTES;
    if (dest_space < nbytes || idx < 0 || idx > nelts) {
        // FIXME: This is not the right way to handle an
        // "insufficient space" error or an "index out of bounds"
        // error.
        return 0;
    }
    // clear all of dest
    // TODO: Is this necessary? Should it be optional?
    memset(dest, 0, dest_space);
    cuda_memcpy_from_device(dest, ptr + idx * hand_impl::SLOT_WIDTH, nbytes);
    return nbytes;
}

template< typename hand_impl >
void
fixnum_array<hand_impl>::retrieve(uint8_t **dest, size_t *dest_len, int idx) const {
    *dest_len = hand_impl::FIXNUM_BYTES;
    *dest = new uint8_t[*dest_len];
    retrieve_into(*dest, *dest_len, idx);
}

template< typename hand_impl >
void
fixnum_array<hand_impl>::retrieve_all(uint8_t **dest, size_t *dest_len, size_t *nelts) const {
    size_t nbytes;
    *nelts = this->nelts;
    nbytes = *nelts * hand_impl::FIXNUM_BYTES;
    *dest = new uint8_t[nbytes];
    // FIXME: This won't correctly zero-pad each element
    memset(dest, 0, nbytes);
    cuda_memcpy_from_device(*dest, ptr, nbytes);
}

template< typename hand_impl >
template< typename Func >
void
fixnum_array<hand_impl>::apply_to_all(Func fn, const fixnum_array<hand_impl> *src, clock_t *t) {
    // TODO: Set this to the number of threads on a single SM on the host GPU.
    constexpr int BLOCK_SIZE = 192;

    // dest and src must be the same length
    assert(nelts == src->nelts);
    // BLOCK_SIZE must be a multiple of warpSize
    static_assert(!(BLOCK_SIZE % WARPSIZE),
            "block size must be a multiple of warpSize");

    // FIXME: Check this calculation
    //int fixnums_per_block = (BLOCK_SIZE / warpSize) * hand_impl::NSLOTS;
    constexpr int fixnums_per_block = BLOCK_SIZE / hand_impl::SLOT_WIDTH;

    // FIXME: nblocks could be too big for a single kernel call to handle
    int nblocks = iceil(src->nelts, fixnums_per_block);

    if (t) *t = clock();
    // nblocks > 0 iff src->nelts > 0
    if (nblocks > 0) {
        hipStream_t stream;
        cuda_check(hipStreamCreate(&stream), "create stream");
        // FIXME: how do I attach the function?
        //stream_attach(stream, fn);
//         cuda_stream_attach_mem(stream, src->ptr);
//         cuda_stream_attach_mem(stream, ptr);
        cuda_check(hipStreamSynchronize(stream), "stream sync");

        dispatch<hand_impl><<< nblocks, BLOCK_SIZE, 0, stream >>>(fn, nelts, ptr, ptr, src->ptr);

        cuda_check(hipPeekAtLastError(), "kernel invocation/run");
        cuda_check(hipStreamSynchronize(stream), "stream sync");
        cuda_check(hipStreamDestroy(stream), "stream destroy");
    }
    if (t) *t = clock() - *t;
}
